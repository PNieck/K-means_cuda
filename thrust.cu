#pragma once

#include "KMeansAlg.cuh"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/fill.h>
#include <thrust/transform.h>
#include <thrust/for_each.h>
#include <thrust/zip_function.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/count.h>
#include <thrust/transform_reduce.h>

#include "hip/hip_runtime.h"


struct ThrustData
{
	int dim_cnt;
	int points_cnt;
	int centr_cnt;

	thrust::device_vector<float>* points_coord;
	thrust::device_vector<int> centr_indexes;

	thrust::device_vector<int> new_centr_indexes;
	thrust::device_vector<float> min_dist;
	thrust::device_vector<float> act_dist;

	thrust::host_vector<float>* centr_coord;
};


ThrustData createThrustData(const Points& points, const Centroids& centroids)
{
	ThrustData result;

	result.dim_cnt = points.dim_cnt;
	result.points_cnt = points.cnt;
	result.centr_cnt = centroids.cnt;

	result.points_coord = new thrust::device_vector<float>[result.dim_cnt];
	for (int i = 0; i < result.dim_cnt; i++) {
		result.points_coord[i] = thrust::device_vector<float>(points.coordinates[i], points.coordinates[i] + points.cnt);
	}

	result.centr_indexes = thrust::device_vector<int>(points.cnt, -1);

	result.new_centr_indexes = thrust::device_vector<int>(points.cnt);

	result.min_dist = thrust::device_vector<float>(points.cnt);
	result.act_dist = thrust::device_vector<float>(points.cnt);

	result.centr_coord = new thrust::host_vector<float>[result.dim_cnt];
	for (int i = 0; i < result.dim_cnt; i++) {
		result.centr_coord[i] = thrust::host_vector<float>(centroids.coordinates[i], centroids.coordinates[i] + centroids.cnt);
	}

	return result;
}


struct distance_functor
{
	const float centr_coord;

	distance_functor(float coord) : centr_coord(coord) {}

	__host__ __device__
		float operator()(const float& point_coord, const float& dist) const {
		float temp = point_coord - centr_coord;
		return dist + temp * temp;
	}
};


void calculate_dist(ThrustData& data, int centroid_index) {
	thrust::fill(data.act_dist.begin(), data.act_dist.end(), 0);

	for (int j = 0; j < data.dim_cnt; j++) {
		thrust::transform(data.points_coord[j].begin(), data.points_coord[j].end(), data.act_dist.begin(), data.act_dist.begin(), distance_functor(data.centr_coord[j][centroid_index]));
	}
}


struct update_min_dist_functor
{
	const int act_centoid;

	update_min_dist_functor(int _act_centroid) : act_centoid(_act_centroid) {}

	__host__ __device__
		void operator()(float& min_dist, const float& act_dist, int& new_centroid_index)
	{
		if (min_dist > act_dist) {
			min_dist = act_dist;
			new_centroid_index = act_centoid;
		}
	}
};


struct count_new_centroids_functor
{
	__host__ __device__
		bool operator()(const thrust::tuple<int, int>& tuple)
	{
		if (thrust::get<0>(tuple) != thrust::get<1>(tuple)) {
			return true;
		}

		return false;
	}
};


int find_nearest_centroids(ThrustData& data)
{
	thrust::fill(data.min_dist.begin(), data.min_dist.end(), std::numeric_limits<float>::infinity());

	for (int i = 0; i < data.centr_cnt; i++) {
		calculate_dist(data, i);

		thrust::for_each(thrust::make_zip_iterator(data.min_dist.begin(), data.act_dist.begin(), data.new_centr_indexes.begin()),
						 thrust::make_zip_iterator(data.min_dist.end(),   data.act_dist.end(),   data.new_centr_indexes.end()  ),
						 thrust::make_zip_function(update_min_dist_functor(i)));
	}

	int result = thrust::count_if(thrust::make_zip_iterator(data.centr_indexes.begin(), data.new_centr_indexes.begin()),
								  thrust::make_zip_iterator(data.centr_indexes.end(),   data.new_centr_indexes.end()),
								  count_new_centroids_functor());

	thrust::swap(data.centr_indexes, data.new_centr_indexes);

	return result;
}


struct equal_functor
{
	const int val;

	equal_functor(int _val) : val(_val) {}

	__host__ __device__
		float operator()(const thrust::tuple<float, int>& tuple)
	{
		if (val == thrust::get<1>(tuple))
			return thrust::get<0>(tuple);

		return 0;
	}
};


void recalculate_centroids(ThrustData& data)
{
	for (int i = 0; i < data.centr_cnt; i++) {
		int elems = thrust::count(data.centr_indexes.begin(), data.centr_indexes.end(), i);

		for (int j = 0; j < data.dim_cnt; j++) {
			 float val = thrust::transform_reduce(thrust::make_zip_iterator(data.points_coord[j].begin(), data.centr_indexes.begin()),
												  thrust::make_zip_iterator(data.points_coord[j].end(), data.centr_indexes.end()),
												  equal_functor(i),
												  0,
												  thrust::plus<float>());
			 if (elems == 0) {
				 data.centr_coord[j][i] = std::numeric_limits<float>::infinity();
			 }
			 else {
				 data.centr_coord[j][i] = val / elems;
			 }
		}
	}
}


void create_result(const ThrustData& data, Points& points, Centroids& centroids)
{
	thrust::copy(data.centr_indexes.begin(), data.centr_indexes.end(), points.centroids_indexes);

	for (int i = 0; i < data.dim_cnt; i++) {
		thrust::copy(data.centr_coord[i].begin(), data.centr_coord[i].end(), centroids.coordinates[i]);
	}
}


int KMeansAlg::thrust_version(Points& points, Centroids& centroids, float threshold, int max_it)
{
	ThrustData data = createThrustData(points, centroids);

	int iterations = 0;
	int cent_changes = points.cnt;

	while ((float)cent_changes / (float)points.cnt > threshold && iterations < max_it) {
		cent_changes = find_nearest_centroids(data);

		recalculate_centroids(data);
		iterations++;
	}

	create_result(data, points, centroids);

	return iterations;
}
